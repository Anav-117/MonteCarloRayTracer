#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Defines.cuh"
#include "Utils.cuh"
#include "Vec3.cuh"
#include "Ray.cuh"
#include "Color.cuh"
#include "Sphere.cuh"
#include "Hittable.cuh"
#include "Camera.cuh"
#include "Material.cuh"

#include <iostream>
#include <fstream>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

DEV bool lambertian_scatter(const ray& r_in, const hit_record& rec, color& attenuation, ray& scattered, color albedo, vec3 random) {
    auto scatter_direction = rec.normal + random;
    
    // Catch degenerate scatter direction
    if (scatter_direction.near_zero())
        scatter_direction = rec.normal;
    
    scattered = ray(rec.p, scatter_direction);
    attenuation = albedo;
    return true;
}

DEV bool metal_scatter(const ray& r_in, const hit_record& rec, color& attenuation, ray& scattered, color albedo, double fuzz, vec3 random) {
    vec3 reflected = reflect(unit_vector(r_in.direction()), rec.normal);
    scattered = ray(rec.p, reflected + fuzz * random);
    attenuation = albedo;
    return (dot(scattered.direction(), rec.normal) > 0);
}

DEV bool dielectric_scatter(const ray& r_in, const hit_record& rec, color& attenuation, ray& scattered, double ir, double random) {
    attenuation = color(1.0, 1.0, 1.0);
    double refraction_ratio = rec.front_face ? (1.0 / ir) : ir;

    vec3 unit_direction = unit_vector(r_in.direction());
    double cos_theta = fmin(dot(-unit_direction, rec.normal), 1.0);
    double sin_theta = sqrt(1.0 - cos_theta * cos_theta);

    bool cannot_refract = refraction_ratio * sin_theta > 1.0;
    vec3 direction;

    if (cannot_refract || reflectance(cos_theta, refraction_ratio) > random)
        direction = reflect(unit_direction, rec.normal);
    else
        direction = refract(unit_direction, rec.normal, refraction_ratio);

    scattered = ray(rec.p, direction);
    return true;
}

__global__ void ray_color(vec3* KRNG_Diffuse, ray* r, camera* cam, color* Image, sphere* world, int samples_per_pixel, int image_width, int image_height, int world_size, int max_depth) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;

    hit_record rec;

    ray R = r[j];

    int num_hits = 0;
    color FinalColor = color(0, 0, 0);
    color FinalAttenuation = color(0,0,0);

    int depth = max_depth;
    //printf("Here1\n");

    while (depth > 0) {

        hit_record temp_rec;
        bool hit_anything = false;
        double tmax = infinity;
        double tmin = 0.001;
        double closest_so_far = tmax;

        for (int i = 0; i < world_size; i++) {
            if (world[i].hit(R, tmin, closest_so_far, temp_rec)) {
                hit_anything = true;
                closest_so_far = temp_rec.t;
                rec = temp_rec;
            }
        }

        if (hit_anything) {
            num_hits++;
            ray scattered;
            color attenuation;
            if (rec.Mat.material == 0) {
                if (lambertian_scatter(R, rec, attenuation, scattered, rec.Mat.albedo, KRNG_Diffuse[j * max_depth + (depth-1)])) {
                    if (FinalAttenuation == color(0, 0, 0)) {
                        FinalAttenuation = attenuation;
                    }
                    else {
                        FinalAttenuation = FinalAttenuation * attenuation;
                    }
                }
            }
            else if (rec.Mat.material == 1) {
                if (metal_scatter(R, rec, attenuation, scattered, rec.Mat.albedo, rec.Mat.fuzz, KRNG_Diffuse[j * max_depth + (depth - 1)])) {
                    if (FinalAttenuation == color(0, 0, 0)) {
                        FinalAttenuation = attenuation;
                    }
                    else {
                        FinalAttenuation = FinalAttenuation * attenuation;
                    }
                }
            }
            else if (rec.Mat.material == 2) {
                if (dielectric_scatter(R, rec, attenuation, scattered, rec.Mat.ir, KRNG_Diffuse[j * max_depth + (depth - 1)].x())) {
                    if (FinalAttenuation == color(0, 0, 0)) {
                        FinalAttenuation = attenuation;
                    }
                    else {
                        FinalAttenuation = FinalAttenuation * attenuation;
                    }
                }
            }
            R = scattered;
            depth--;
        }
        else {
            vec3 unit_direction = unit_vector(R.direction());
            auto t = 0.5 * (unit_direction.y() + 1.0);
            FinalColor = (1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0);
            break;
        }

    }

    if (!(FinalAttenuation == color(0, 0, 0))) {
        FinalColor = FinalColor * FinalAttenuation;
    }

    Image[j] = FinalColor;

    //printf("Completed Pixel - %d\n", j);

}

int main() {
	// Image
	const auto aspect_ratio = 16.0 / 9.0;
	const int image_width = 400;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    const int samples_per_pixel = 500;
    const int max_depth = 50;

    // Image File

    std::ofstream ImageFile;
    ImageFile.open("Image.ppm");

    // World
    int world_size = 4;

    sphere* world = new sphere[world_size];

    material ground_material = { 0, color(0.5, 0.5, 0.5), 0, 0 }; //lambertian
    world[0] = sphere(point3(0, -1000, 0), 1000, ground_material);

    material material1 = { 2, color(0,0,0), 0, 1.5 };
    world[1] = sphere(point3(0, 1, 0), -1.0, material1);

    material material2 = { 0, color(0.4, 0.2, 0.1), 0, 0 };
    world[2] = sphere(point3(-1, 1, 0), 1.0, material2);

    material material3 = { 1, color(0.7, 0.6, 0.5), 0.0, 0 };
    world[3] = sphere(point3(1, 1, 0), 1.0, material3);
    
    // Camera

    point3 lookfrom(-5, 5, 1);
    point3 lookat(0, 1, 0);
    vec3 vup(0, 1, 0);
    auto dist_to_focus = (lookfrom - lookat).length();
    auto aperture = 0.3;

    camera cam(lookfrom, lookat, vup, 50, aspect_ratio, aperture, dist_to_focus);

    // Generate Rays

    ImageFile << "P3\n" << image_width << " " << image_height << "\n255\n";

    for (int j = image_height - 1; j >= 0; --j) {
        std::cout << "\rScanlines remaining: " << j << ' ' << std::flush;
        for (int i = 0; i < image_width; ++i) {
            ray* Ray_Mat = new ray[samples_per_pixel];
            color* Image = new color[samples_per_pixel];

            for (int s = 0; s < samples_per_pixel; ++s) {
                auto u = (i + random_double()) / (image_width - 1);
                auto v = (j + random_double()) / (image_height - 1);

                ray R = cam.get_ray(u, v);

                Ray_Mat[s] = R;

                Image[s] = color(0.0, 0.0, 0.0);
            }

            vec3* RNG_Diffuse = new vec3[samples_per_pixel * max_depth];
            for (int i = 0; i < samples_per_pixel * max_depth; i++) {
                RNG_Diffuse[i] = random_in_unit_sphere();
            }

            color PixelColor = color(0, 0, 0);

            ray* KRay_Mat = NULL;
            color* KImage = NULL;
            sphere* KWorld = NULL;
            camera* KCam = NULL;
            vec3* KRNG_Diffuse = NULL;
            hipMalloc(&KRNG_Diffuse, samples_per_pixel * max_depth * sizeof(vec3));
            hipMalloc(&KRay_Mat, samples_per_pixel * sizeof(ray));
            hipMalloc(&KImage, samples_per_pixel * sizeof(color));
            hipMalloc(&KWorld, world_size * sizeof(sphere));
            hipMalloc(&KCam, sizeof(camera));
            hipMemcpy(KRNG_Diffuse, RNG_Diffuse, samples_per_pixel * max_depth * sizeof(vec3), hipMemcpyHostToDevice);
            hipMemcpy(KRay_Mat, Ray_Mat, samples_per_pixel * sizeof(ray), hipMemcpyHostToDevice);
            hipMemcpy(KImage, Image, samples_per_pixel * sizeof(color), hipMemcpyHostToDevice);
            hipMemcpy(KWorld, world, world_size * sizeof(sphere), hipMemcpyHostToDevice);
            hipMemcpy(KCam, &cam, sizeof(camera), hipMemcpyHostToDevice);

            ray_color <<<1, samples_per_pixel>>> (KRNG_Diffuse, KRay_Mat, KCam, KImage, KWorld, samples_per_pixel, image_width, image_height, world_size, max_depth);
            hipDeviceSynchronize();

            hipMemcpy(Image, KImage, samples_per_pixel * sizeof(color), hipMemcpyDeviceToHost);

            //HandleCudaKernelError(hipGetLastError());

            for (int s = 0; s < samples_per_pixel; ++s) {
                PixelColor += Image[s];
            }

            write_color(ImageFile, PixelColor, samples_per_pixel);

            delete[] Ray_Mat;
            delete[] Image;
            delete[] RNG_Diffuse;

            hipFree(KRay_Mat);
            hipFree(KImage);
            hipFree(KWorld);
            hipFree(KCam);
            hipFree(KRNG_Diffuse);
        }
    }

    ImageFile.close();

	return 0;
}